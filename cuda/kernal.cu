#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */


#ifndef _BICUBICTEXTURE_CU_
#define _BICUBICTEXTURE_CU_

#include <cfloat>

#include <stdlib.h>
#include <stdio.h>
#include <string.h>

// My Files
#include "vec3.h"
#include "sphere.h"
#include "Ray.h"
#include "hitable_list.h"
#include "hitable.h"
#include "random"

#include <hip/hip_vector_types.h>

// includes, cuda
#include <hip/hip_runtime_api.h>

#define PARTICLE_COUNT 50

sphere spheres[PARTICLE_COUNT];
uint threadPerBlock = 25;
uint blocks = PARTICLE_COUNT / threadPerBlock;

typedef unsigned int uint;
typedef unsigned char uchar;

__device__ vec3 castRay(const ray &r, hitable **world) {
    hit_record rec;
    if ((*world)->hit(r, 0.0, FLT_MAX, rec)) {
        return 0.5f * vec3(rec.normal.x() + 1.0f, rec.normal.y() + 1.0f, rec.normal.z() + 1.0f);
    } else {
//        vec3 unit_direction = unit_vector(r.direction());
//        float t = 0.5f * (unit_direction.y() + 1.0f);
//        return (1.0f - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
        return vec3(0, 0, 0);
    }
}

__global__ void d_render(uchar4 *d_output, uint width, uint height, hitable **d_world) {
    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;
    uint i = y * width + x;
    float u = x / (float) width; //----> [0, 1]x[0, 1]
    float v = y / (float) height;
    u = 2.0 * u - 1.0; //---> [-1, 1]x[-1, 1]
    v = -(2.0 * v - 1.0);
    u *= width / (float) height;
    u *= 2.0;
    v *= 2.0;
    vec3 eye = vec3(0, 0.5, 1.5);
    float distFrEye2Img = 1.0;;
    if ((x < width) && (y < height)) {
        //for each pixel
        vec3 pixelPos = vec3(u, v, eye.z() - distFrEye2Img);
        //fire a ray:
        ray r;
        r.O = eye;
        r.Dir = pixelPos - eye;
        //view direction along negtive z-axis!
        vec3 col = castRay(r, d_world);
        float red = col.x();
        float green = col.y();
        float blue = col.z();
        d_output[i] = make_uchar4(red * 255, green * 255, blue * 255, 0);
    }

}

extern "C"
void onIdle() {
    for (auto &sphere : spheres) {
        sphere.move(0.005);
    }
}

__global__ void create_world(sphere *d_spheres, hitable **d_list, hitable **d_world) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {

//    int i = threadIdx.x + (blockDim.x * blockIdx.x);

        for (int i = 0; i < PARTICLE_COUNT; i++) {
            auto s = d_spheres[i];
//            *(d_list + i) = &d_spheres[i];

//            printf("%f\n", s.position.x());

            *(d_list + i) = new sphere(s);
        }

        *d_world = new hitable_list(d_list, PARTICLE_COUNT);
    }
}

__global__ void free_world(hitable **d_list, hitable **d_world) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        for (int i = 0; i < PARTICLE_COUNT; i++) {
            delete *(d_list + i);
        }
        delete *d_world;
    }
}

__global__ void move_particles(sphere *d_spheres) {
//    int i = threadIdx.x + (blockDim.x * blockIdx.x);
//    d_spheres[i].move(1);
}

extern "C"
void render(int width, int height, dim3 blockSize, dim3 gridSize, uchar4 *output) {
    sphere *d_particleList;
    checkCudaErrors(hipMalloc((void **) &d_particleList, PARTICLE_COUNT * sizeof(sphere)));
    checkCudaErrors(hipMemcpy(d_particleList, spheres, PARTICLE_COUNT * sizeof(sphere), hipMemcpyHostToDevice));

    // make our world of hitables
    hitable **d_list;
    checkCudaErrors(hipMalloc((void **) &d_list, 2 * sizeof(hitable *)));
    hitable **d_world;
    checkCudaErrors(hipMalloc((void **) &d_world, sizeof(hitable *)));

    create_world <<< blocks, threadPerBlock >>>(d_particleList, d_list, d_world);

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

//    move_particles <<< blocks, threadPerBlock >>>(d_particleList);
//
//    checkCudaErrors(hipGetLastError());
//    checkCudaErrors(hipDeviceSynchronize());

    // call CUDA kernel, writing results to PBO memory
    d_render <<< gridSize, blockSize >>>(output, width, height, d_world);
    getLastCudaError("kernel failed");

    free_world<<< blocks, threadPerBlock >>>(d_list, d_world);
    getLastCudaError("kernel failed");

    checkCudaErrors(hipMemcpy(spheres, d_particleList, PARTICLE_COUNT * sizeof(sphere), hipMemcpyDeviceToHost));
}

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << "at " << file << ":" << line << " '"
                  << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

#endif
